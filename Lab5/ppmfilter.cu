#include "hip/hip_runtime.h"

#include <stdio.h>
#include "readppm.c"
#ifdef __APPLE__
	#include <GLUT/glut.h>
	#include <OpenGL/gl.h>
#else
	#include <GL/glut.h>
#endif

#define BLOCKDIM 32

__device__ void setPixel(unsigned char* out, unsigned char* gpu_img, unsigned char* source, int local_index, int global_index){
  gpu_img[local_index + 0] = source[global_index + 0];
  gpu_img[local_index + 1] = source[global_index + 1];
  gpu_img[local_index + 2] = source[global_index + 2];
 /* out[global_index+0] = 255;
	out[global_index+1] = 0;
	out[global_index+2] = 0;*/
}

__global__ void filter(unsigned char *image, unsigned char *out, int m, int n)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x; //x
	int y = blockIdx.y * blockDim.y + threadIdx.y; //y
	int yy = threadIdx.y+2; //yy
	int xx = threadIdx.x+2; //xx
	int nn = blockDim.x+4;
	int sumx, sumy, sumz, k, l;

// printf is OK under -arch=sm_20 
// printf("%d %d %d %d\n", y, x, n, m);

	__shared__ unsigned char gpu_image [(BLOCKDIM+4) * (BLOCKDIM+4) * 3];


		if (yy <= 3 && y > 1){ // upper border
		  if(xx <= 3 && x > 1){ // left border
		      setPixel(out, gpu_image, image, ((yy-2)*nn+(xx-2))*3, ((y-2)*n+(x-2))*3);
		  } else if (xx >= nn-4 && x < n-2){ // right border
		      setPixel(out, gpu_image, image, ((yy-2)*nn+(xx+2))*3, ((y-2)*n+(x+2))*3);
		  }
		  setPixel(out, gpu_image, image, ((yy-2)*nn+xx)*3, ((y-2)*n+x)*3);
		  
		} else if(yy >= nn-4 && y < n-2){ // lower border
		  if(xx <= 3 && x > 1){ // left border
		      setPixel(out, gpu_image, image, ((yy+2)*nn+(xx-2))*3, ((y+2)*n+(x-2))*3);
		  } else if (xx >= nn-4 && x < n-2){ // right border
		      setPixel(out, gpu_image, image, ((yy+2)*nn+(xx+2))*3, ((y+2)*n+(x+2))*3);
		  }
		  setPixel(out, gpu_image, image, ((yy+2)*nn+xx)*3, ((y+2)*n+x)*3);

		}

		if(xx <= 3 && x > 1){ // left border
		  setPixel(out, gpu_image, image, (yy*nn+xx-2)*3, (y*n+x-2)*3);
		} else if(xx >= nn-4 && x < n-2){ // right border
		  setPixel(out, gpu_image, image, (yy*nn+xx+2)*3, (y*n+x+2)*3);
		}
		
		
	  setPixel(out, gpu_image, image, (yy*nn+xx)*3, (y*n+x)*3);

	__syncthreads();
	
		// Filter kernel
		sumx=0;sumy=0;sumz=0;
		for(k=-2;k<3;k++)
			for(l=-2;l<3;l++)
			{
				sumx += gpu_image[((yy + k) * nn + (xx + l)) * 3 + 0];
				sumy += gpu_image[((yy + k) * nn + (xx + l)) * 3 + 1];
				sumz += gpu_image[((yy + k) * nn + (xx + l)) * 3 + 2];
			}
		out[(y*n+x)*3+0] = sumx/25;
		out[(y*n+x)*3+1] = sumy/25;
		out[(y*n+x)*3+2] = sumz/25;
 		
	//__syncthreads();

}

__global__ void filterNaive(unsigned char *image, unsigned char *out, int n, int m)
{
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        int j = blockIdx.y * blockDim.y + threadIdx.y;
        int sumx, sumy, sumz, k, l;

// printf is OK under --device-emulation
//        printf("%d %d %d %d\n", i, j, n, m);

        if (j < n && i < m)
        {
                out[(i*n+j)*3+0] = image[(i*n+j)*3+0];
                out[(i*n+j)*3+1] = image[(i*n+j)*3+1];
                out[(i*n+j)*3+2] = image[(i*n+j)*3+2];
        }
        
        if (i > 1 && i < m-2 && j > 1 && j < n-2)
                {
                        // Filter kernel
                        sumx=0;sumy=0;sumz=0;
                        for(k=-2;k<3;k++)
                                for(l=-2;l<3;l++)
                                {
                                        sumx += image[((i+k)*n+(j+l))*3+0];
                                        sumy += image[((i+k)*n+(j+l))*3+1];
                                        sumz += image[((i+k)*n+(j+l))*3+2];
                                }
                        out[(i*n+j)*3+0] = sumx/25;
                        out[(i*n+j)*3+1] = sumy/25;
                        out[(i*n+j)*3+2] = sumz/25;
                }
}


// Compute CUDA kernel and display image
void Draw()
{
	unsigned char *image, *out;
	int n, m;
	unsigned char *dev_image, *dev_out;
	
	image = readppm("maskros512.ppm", &n, &m);
	out = (unsigned char*) malloc(n*m*3);
	
	hipMalloc( (void**)&dev_image, n*m*3);
	hipMalloc( (void**)&dev_out, n*m*3);
	hipMemcpy( dev_image, image, n*m*3, hipMemcpyHostToDevice);
	
	dim3 dimBlock( BLOCKDIM, BLOCKDIM );
	dim3 dimGrid( 512/BLOCKDIM, 512/BLOCKDIM );
	//dim3 dimBlock( 512/8 , 512/8);
	//dim3 dimGrid( 8, 8 );
	

  hipEvent_t myEvent, myEvent2;
  hipEventCreate(&myEvent);
  hipEventCreate(&myEvent2);
  hipEventRecord(myEvent, 0);
  hipEventSynchronize(myEvent);


	filter<<<dimGrid, dimBlock>>>(dev_image, dev_out, n, m);
	hipDeviceSynchronize();

  hipEventRecord(myEvent2, 0);
  hipEventSynchronize(myEvent2);
	float theTime;
  hipEventElapsedTime(&theTime, myEvent, myEvent2);
	printf("The gpu calculation (optimized) took: %0.2f ms\n", theTime);


  hipEventCreate(&myEvent);
  hipEventCreate(&myEvent2);
  hipEventRecord(myEvent, 0);
  hipEventSynchronize(myEvent);
	filterNaive<<<dimGrid, dimBlock>>>(dev_image, dev_out, n, m);
	hipDeviceSynchronize();

  hipEventRecord(myEvent2, 0);
  hipEventSynchronize(myEvent2);
  hipEventElapsedTime(&theTime, myEvent, myEvent2);
	printf("The gpu calculation (naive) took: %0.2f ms\n", theTime);


	hipMemcpy( out, dev_out, n*m*3, hipMemcpyDeviceToHost );
	hipFree(dev_image);
	hipFree(dev_out);
	
// Dump the whole picture onto the screen.	
	glClearColor( 0.0, 0.0, 0.0, 1.0 );
	glClear( GL_COLOR_BUFFER_BIT );
	glRasterPos2f(-1, -1);
	glDrawPixels( n, m, GL_RGB, GL_UNSIGNED_BYTE, image );
	glRasterPos2i(0, -1);
	glDrawPixels( n, m, GL_RGB, GL_UNSIGNED_BYTE, out );
	glFlush();

}

// Main program, inits
int main( int argc, char** argv) 
{
	glutInit(&argc, argv);
	glutInitDisplayMode( GLUT_SINGLE | GLUT_RGBA );
	glutInitWindowSize( 1024, 512 );
	glutCreateWindow("CUDA on live GL");
	glutDisplayFunc(Draw);
	
	glutMainLoop();
}
