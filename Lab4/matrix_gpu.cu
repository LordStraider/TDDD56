#include "hip/hip_runtime.h"
// Matrix addition, CPU version
// gcc matrix_cpu.c -o matrix_cpu -std=c99

#include <stdio.h>
#include "milli.h"


const int N = 1024;
int grids = 64;
int blocksize = N / grids; 


__global__ 
void matrix(float *a, float *b, float *c) 
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int index = x * N + y;
	c[index] = a[index] + b[index];
}

void add_matrix(float *a, float *b, float *c, int N)
{
	int index;
	
	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
		{
			index = i + j*N;
			c[index] = a[index] + b[index];
		}
}

int main()
{
	const int size = N*N*sizeof(float);
	
	float *a = new float[N*N];
	float *b = new float[N*N];
	float *c = new float[N*N];

	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
		{
			a[i+j*N] = 10 + i;
			b[i+j*N] = (float)j / N;
		}
	
	ResetMilli();
	add_matrix(a, b, c, N);	
	int time = GetMilliseconds();
	/*for (int i = 0; i < N; i++)	{
		for (int j = 0; j < N; j++)	{
			printf("%0.2f ", c[i+j*N]);
			c[i+j*N] = -1;
		}
		printf("\n");
	}*/
	printf("cpu took: %d ms", time);
	printf("\n-----------------\n");

	float *cd, *a_g, *b_g;
	hipMalloc( (void**)&cd, size );
	hipMalloc( (void**)&a_g, size );
	hipMalloc( (void**)&b_g, size );
	hipMemcpy( a_g, a, size, hipMemcpyHostToDevice ); 
	hipMemcpy( b_g, b, size, hipMemcpyHostToDevice ); 
	hipMemcpy( cd, c, size, hipMemcpyHostToDevice ); 
	dim3 dimBlock( blocksize, blocksize);
	dim3 dimGrid( grids, grids );

  hipEvent_t myEvent, myEvent2;
  hipEventCreate(&myEvent);
  hipEventCreate(&myEvent2);

  hipEventRecord(myEvent, 0);
  hipEventSynchronize(myEvent);

	matrix<<<dimGrid, dimBlock>>>(a_g, b_g, cd);
	hipDeviceSynchronize();


  hipEventRecord(myEvent2, 0);
  hipEventSynchronize(myEvent2);

	float theTime;

  hipEventElapsedTime(&theTime, myEvent, myEvent2);

	hipMemcpy( c, cd, size, hipMemcpyDeviceToHost ); 
	hipFree( cd );
	hipFree( a_g );
	hipFree( b_g );
	
	/*for (int i = 0; i < N; i++)	{
		for (int j = 0; j < N; j++)	{
			printf("%0.2f ", c[i+j*N]);
		}
		printf("\n");
	}*/

	printf("The gpu calculation took: %0.2f ms\n", theTime);
}
